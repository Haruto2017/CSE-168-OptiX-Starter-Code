#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_device.h>
#include <optixu/optixu_math_namespace.h>
#include "random.h"

#include "Payloads.h"
#include "Geometries.h"
#include "Light.h"

using namespace optix;

#define PI 3.1415926538

// Declare light buffers
rtBuffer<ParallelogramLight> qlights;
rtBuffer<PointLight> plights;
rtBuffer<DirectionalLight> dlights;

// Declare variables
rtDeclareVariable(Payload, payload, rtPayload, );
rtDeclareVariable(rtObject, root, , );

// Declare attibutes 
rtDeclareVariable(Attributes, attrib, attribute attrib, );
rtDeclareVariable(float3, intersection, attribute Intersection, );
rtDeclareVariable(float3, normal, attribute Normal, );
rtDeclareVariable(float3, view, attribute View, );

RT_PROGRAM void closestHit()
{
    // TDOO: calculate the color using the Blinn-Phong reflection model
    float3 result = attrib.emission + attrib.ambient;
    float epsilon = 0.001f;

    for (int i = 0; i < qlights.size(); ++i)
    {
        ParallelogramLight qlight = qlights[i];
        float3 u0 = qlight.a - intersection;
        float3 u1 = qlight.a + qlight.ab - intersection;
        float3 u2 = qlight.a + qlight.ab + qlight.ac - intersection;
        float3 u3 = qlight.a + qlight.ac - intersection;
        float theta0 = acos(dot(normalize(u0), normalize(u1)));
        float3 reverseL0 = normalize(cross(u0, u1));
        float theta1 = acos(dot(normalize(u1), normalize(u2)));
        float3 reverseL1 = normalize(cross(u1, u2));
        float theta2 = acos(dot(normalize(u2), normalize(u3)));
        float3 reverseL2 = normalize(cross(u2, u3));
        float theta3 = acos(dot(normalize(u3), normalize(u0)));
        float3 reverseL3 = normalize(cross(u3, u0));
        float3 irradiance = 0.5 * (theta0 * reverseL0 + theta1 * reverseL1 + theta2 * reverseL2 + theta3 * reverseL3);
        result += (attrib.diffuse / PI) * qlight.intensity * dot(irradiance, normal);
    }

    //payload.weight /= (1.0 + 0.1 * dis + 0.05 * dis * dis);
    payload.radiance = result;
}