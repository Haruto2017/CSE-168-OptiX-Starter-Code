#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_device.h>
#include <optixu/optixu_math_namespace.h>
#include "random.h"

#include "Payloads.h"
#include "Geometries.h"
#include "Light.h"

using namespace optix;

#define PI 3.1415926538

// Declare light buffers
rtBuffer<ParallelogramLight> qlights;
rtBuffer<PointLight> plights;
rtBuffer<DirectionalLight> dlights;

// Declare variables
rtDeclareVariable(Payload, payload, rtPayload, );
rtDeclareVariable(rtObject, root, , );
rtDeclareVariable(uint, lightsamples, , );
rtDeclareVariable(uint, lightstratify, , );

// Declare attibutes 
rtDeclareVariable(Attributes, attrib, attribute attrib, );
rtDeclareVariable(float3, intersection, attribute Intersection, );
rtDeclareVariable(float3, normal, attribute Normal, );
rtDeclareVariable(float3, view, attribute View, );

RT_PROGRAM void closestHit()
{
    // TDOO: calculate the color using the Blinn-Phong reflection model
    float3 result = attrib.emission + attrib.ambient;
    float epsilon = 0.001f;
    
    for (int i = 0; i < qlights.size(); ++i)
    {
        ParallelogramLight qlight = qlights[i];
        float3 sum = make_float3(0, 0, 0);
        for (int j = 0; j < lightsamples; ++j)
        {
            float u1;
            float u2;
            if (lightstratify == 1)
            {
                uint size = (uint)sqrt((float)lightsamples);
                u1 = rnd(payload.seed) / size;
                u2 = rnd(payload.seed) / size;
                u1 += ((float)(j / size)) / size;
                u2 += ((float)(j % size)) / size;
            }
            else
            {
                u1 = rnd(payload.seed);
                u2 = rnd(payload.seed);
            }
            //rtPrintf("%f %f \n", u1, u2);
            float3 xl = qlight.a + u1 * qlight.ab + u2 * qlight.ac;
            float3 l = normalize(xl - intersection);
            Ray shadowRay = make_Ray(intersection, l, 1, epsilon, length(xl - intersection) - epsilon);
            ShadowPayload shadowPayload;
            shadowPayload.isVisible = true;
            rtTrace(root, shadowRay, shadowPayload);
            if (!shadowPayload.isVisible)
            {
                continue;
            }
            float3 r = 2 * normal * dot(normal, -view) + view;
            float3 brdf = attrib.diffuse / PI + attrib.specular * ((attrib.shininess + 2) / (2 * PI)) * pow(fmaxf(0.0, dot(r, l)), attrib.shininess);
            float costhetai = fmaxf(0.0, dot(normal, l));
            float3 nl = normalize(cross(qlight.ab, qlight.ac));
            float costhetao = fmaxf(0.0, dot(l, nl));
            float g = (costhetai * costhetao) / (length(xl - intersection) * length(xl - intersection));
            sum += brdf * g;
        }
        float A = length(cross(qlight.ab, qlight.ac));
        result += qlight.intensity * A * (sum / lightsamples);
    }

    //payload.weight /= (1.0 + 0.1 * dis + 0.05 * dis * dis);
    payload.radiance = result;
}