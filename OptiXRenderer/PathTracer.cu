#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_device.h>
#include <optixu/optixu_math_namespace.h>
#include "random.h"

#include "Payloads.h"
#include "Geometries.h"
#include "Light.h"

using namespace optix;

#define PI 3.1415926538

// Declare light buffers
rtBuffer<ParallelogramLight> qlights;
rtBuffer<PointLight> plights;
rtBuffer<DirectionalLight> dlights;

// Declare variables
rtDeclareVariable(Payload, payload, rtPayload, );
rtDeclareVariable(rtObject, root, , );
rtDeclareVariable(uint, lightsamples, , );
rtDeclareVariable(uint, lightstratify, , );
rtDeclareVariable(uint, importancesampling, , );

// Declare attibutes 
rtDeclareVariable(Attributes, attrib, attribute attrib, );
rtDeclareVariable(float3, intersection, attribute Intersection, );
rtDeclareVariable(float3, normal, attribute Normal, );
rtDeclareVariable(float3, view, attribute View, );
rtDeclareVariable(float, area, attribute Area, );

RT_PROGRAM void closestHit()
{
    float epsilon = 0.001f;
    float3 result = make_float3(0);
    uint brdf = attrib.brdf;
    //return the radiance if reach a light source (only check once when using next event estimation)
    if (abs(attrib.emission.x - 0.0) > epsilon || abs(attrib.emission.y - 0.0) > epsilon || abs(attrib.emission.z - 0.0) > epsilon)
    {
        if (payload.NEE == 0 || payload.first == 1)
        {
            payload.radiance += (attrib.emission / area) * payload.pathTracingWeight;
        }
        payload.done = true;
        return;
    }

    if (payload.NEE == 1)
    {
        for (int i = 0; i < qlights.size(); ++i)
        {
            ParallelogramLight qlight = qlights[i];
            float3 sum = make_float3(0, 0, 0);
            for (int j = 0; j < lightsamples; ++j)
            {
                float u1;
                float u2;
                if (lightstratify == 1)
                {
                    uint size = (uint)sqrt((float)lightsamples);
                    u1 = rnd(payload.seed) / size;
                    u2 = rnd(payload.seed) / size;
                    u1 += ((float)(j / size)) / size;
                    u2 += ((float)(j % size)) / size;
                }
                else
                {
                    u1 = rnd(payload.seed);
                    u2 = rnd(payload.seed);
                }
                //rtPrintf("%f %f \n", u1, u2);
                float3 xl = qlight.a + u1 * qlight.ab + u2 * qlight.ac;
                float3 l = normalize(xl - intersection);
                Ray shadowRay = make_Ray(intersection, l, 1, epsilon, length(xl - intersection) - epsilon);
                ShadowPayload shadowPayload;
                shadowPayload.isVisible = true;
                rtTrace(root, shadowRay, shadowPayload);
                if (!shadowPayload.isVisible)
                {
                    continue;
                }
                float3 brdf_val;
                if (brdf == 1)
                {
                    float3 r = view - 2 * normal * dot(normal, view);
                    brdf_val = attrib.diffuse / PI + attrib.specular * ((attrib.shininess + 2) / (2 * PI)) * pow(fmaxf(0.0, dot(r, l)), attrib.shininess);
                }
                else
                {
                    if (dot(normal, l) < 0)
                    {
                        brdf_val = make_float3(0);
                    }
                    else
                    {
                        float3 h = normalize(-view + l);
                        //Normal Distribution Function term (GGX)
                        float a_2 = attrib.roughness * attrib.roughness;
                        float theta_h = acos(clamp(dot(h, normal), (float)epsilon, (float)1));
                        float br = pow(a_2 + tan(theta_h) * tan(theta_h), (float)2.0);
                        float bl = PI * pow(clamp(dot(h, normal), (float)epsilon, (float)1), (float)4.0);
                        float D = a_2 / (br * bl);

                        //Shadow Masking term (Smith G Function)
                        float G1_l = 0;
                        if (dot(l, normal) > 0)
                        {
                            float theta_l = acos(dot(l, normal));
                            G1_l = 2 / (1 + sqrt(1 + a_2 * pow(tan(theta_l), (float)2.0)));
                        }
                        float G1_v = 0;
                        if (dot(-view, normal) > 0)
                        {
                            float theta_v = acos(dot(-view, normal));
                            G1_v = 2 / (1 + sqrt(1 + a_2 * pow(tan(theta_v), (float)2.0)));
                        }
                        float G = G1_l * G1_v;

                        //Fresnel term (Schlick's Approximation)
                        float3 F = attrib.specular + (1 - attrib.specular) * pow(1 - clamp(dot(l, h), (float)epsilon, (float)1), (float)5.0);
                        //final output
                        brdf_val = F * G * D / (4 * dot(l, normal) * dot(-view, normal));
                        brdf_val += attrib.diffuse / PI;
                    }
                }
                float costhetai = fmaxf(0.0, dot(normal, l));
                float3 nl = normalize(cross(qlight.ab, qlight.ac));
                float costhetao = fmaxf(0.0, dot(l, nl));
                float g = (costhetai * costhetao) / (length(xl - intersection) * length(xl - intersection));
                sum += brdf_val * g;
            }
            float A = length(cross(qlight.ab, qlight.ac));
            result += qlight.intensity * A * (sum / lightsamples);
        }
    }
    //Russian Roulette
    float boost = 1.0;
    if (payload.RR == 1)
    {
        float curr = rnd(payload.seed);
        float q = 1 - fminf(fmaxf(fmaxf(payload.pathTracingWeight.x, payload.pathTracingWeight.y), payload.pathTracingWeight.z), 1.0);
        if (curr < q)
        {
            payload.done = true;
            payload.radiance += result * payload.pathTracingWeight;
            return;
        }
        else
        {
            boost = 1 / (1 - q);
        }
    }
    //indirect lighting
    float pdf_inver;
    float3 l;
    float3 h;
    //importance sampling the integrated function
    //sample the hemisphere, cosine term, or brdf
    if (importancesampling == 1)
    {
        float u1 = rnd(payload.seed);
        float u2 = rnd(payload.seed);
        float theta = acos(u1);
        float phi = 2 * PI * u2;
        float3 s = make_float3(cos(phi) * sin(theta), sin(theta) * sin(phi), cos(theta));
        float3 a;
        if (abs(normal.y - 1.0) < epsilon)
        {
            a = make_float3(1, 0, 0);
        }
        else
        {
            a = make_float3(0, 1, 0);
        }
        float3 u = normalize(cross(a, normal));
        float3 v = cross(normal, u);
        l = s.x * u + s.y * v + s.z * normal;
        pdf_inver = 2 * PI;
    }
    else if (importancesampling == 2)
    {
        float u1 = rnd(payload.seed);
        float u2 = rnd(payload.seed);
        float theta = acos(u1 * u1);
        float phi = 2 * PI * u2;
        float3 s = make_float3(cos(phi) * sin(theta), sin(theta) * sin(phi), cos(theta));
        float3 a;
        if (abs(normal.y - 1.0) < epsilon)
        {
            a = make_float3(1, 0, 0);
        }
        else
        {
            a = make_float3(0, 1, 0);
        }
        float3 u = normalize(cross(a, normal));
        float3 v = cross(normal, u);
        l = s.x * u + s.y * v + s.z * normal;
        pdf_inver = PI / dot(normal, l);
    }
    else if (importancesampling == 3)
    {
        if (brdf == 1)
        {
            float s_bar = (attrib.specular.x + attrib.specular.y + attrib.specular.z) / 3;
            float d_bar = (attrib.diffuse.x + attrib.diffuse.y + attrib.diffuse.z) / 3;
            float t;
            if (abs(s_bar) < epsilon && abs(d_bar < epsilon))
            {
                t = 1.0;
            }
            else
            {
                t = s_bar / (d_bar + s_bar);
            }
            float u1 = rnd(payload.seed);
            float u2 = rnd(payload.seed);
            float u3 = rnd(payload.seed);
            float theta;
            float3 w;
            float3 r = 2 * normal * dot(normal, -view) + view;
            if (u1 <= t)
            {
                theta = acos(pow(u2, (float)(1.0 / (attrib.shininess + 1))));
                w = r;
            }
            else
            {
                theta = acos(sqrt(u2));
                w = normal;
            }
            float phi = 2 * PI * u3;
            float3 s = make_float3(cos(phi) * sin(theta), sin(theta) * sin(phi), cos(theta));
            float3 a;
            if (abs(w.y - 1.0) < epsilon)
            {
                a = make_float3(1, 0, 0);
            }
            else
            {
                a = make_float3(0, 1, 0);
            }
            float3 u = normalize(cross(a, w));
            float3 v = cross(w, u);
            l = s.x * u + s.y * v + s.z * w;
            if (u1 <= t)
            {
                pdf_inver = t * ((attrib.shininess + 1) / (2 * PI)) * pow(dot(r, l), attrib.shininess);
                pdf_inver = 1 / pdf_inver;
            }
            else
            {
                pdf_inver = (1 - t) * dot(normal, l) / PI;
                pdf_inver = 1 / pdf_inver;
            }
        }
        else
        {
            float s_bar = (attrib.specular.x + attrib.specular.y + attrib.specular.z) / 3;
            float d_bar = (attrib.diffuse.x + attrib.diffuse.y + attrib.diffuse.z) / 3;
            float t;
            if (abs(s_bar) < epsilon && abs(d_bar < epsilon))
            {
                t = 1.0;
            }
            else
            {
                t = fmaxf(0.25, s_bar / (d_bar + s_bar));
            }
            float u1 = rnd(payload.seed);
            float u2 = rnd(payload.seed);
            float u3 = rnd(payload.seed);
            float theta;
            float phi;
            float3 w = normal;
            //float3 r = 2 * normal * dot(normal, -view) + view;
            //choose between the diffuse and the specular term
            if (u1 <= t)
            {
                phi = 2 * PI * u3;
                theta = atan(attrib.roughness * sqrt(u2) / clamp(sqrt(1 - u2), epsilon, (float)1));
            }
            else
            {
                phi = 2 * PI * u3;
                theta = acos(sqrt(u2));
            }
            //rotate to the hemisphere indicated by the surface normal
            float3 s = make_float3(cos(phi) * sin(theta), sin(theta) * sin(phi), cos(theta));
            float3 a;
            if (abs(w.y - 1.0) < epsilon)
            {
                a = make_float3(1, 0, 0);
            }
            else
            {
                a = make_float3(0, 1, 0);
            }
            float3 u = normalize(cross(a, w));
            float3 v = cross(w, u);
            l = s.x * u + s.y * v + s.z * w;
            h = normalize(-view + l);
            //we only found the half vector for the microfacet brdf so we need to reflect our view vector off the half vector
            if (u1 <= t)
            {
                h = make_float3(l.x, l.y, l.z);
                l = 2 * h * dot(h, -view) + view;
                if (dot(l, normal) < 0)
                {
                    payload.done = true;
                    pdf_inver = 0;
                }
                else
                {
                    float a_2 = attrib.roughness * attrib.roughness;
                    float cos_h = clamp(dot(h, normal), (float)epsilon, (float)1);
                    float theta_h = acos(cos_h);
                    float br = pow(a_2 + tan(theta_h) * tan(theta_h), (float)2.0);
                    float bl = PI * pow(cos(theta_h), (float)4.0);
                    float D = a_2 / (br * bl);

                    pdf_inver = t * cos_h * D / (4 * clamp(dot(h, l), epsilon, (float)1));
                    pdf_inver = 1 / pdf_inver;
                }
            }
            else
            {
                pdf_inver = (1 - t) * dot(normal, l) / PI;
                pdf_inver = 1 / pdf_inver;
            }
        }
    }
    float3 brdf_val;
    if (brdf == 1)
    {
        float3 r = view - 2 * normal * dot(normal, view);
        brdf_val = attrib.diffuse / PI + attrib.specular * ((attrib.shininess + 2) / (2 * PI)) * pow(fmaxf(0.0, dot(r, l)), attrib.shininess);
    }
    else
    {
        if (dot(l, normal) < 0)
        {
            //rtPrintf("any\n");
            brdf_val = make_float3(0);
        }
        else
        {
            //Normal Distribution Function term (GGX)
            float a_2 = attrib.roughness * attrib.roughness;
            float theta_h = acos(clamp(dot(h, normal), (float)epsilon, (float)1));
            float br = pow(a_2 + tan(theta_h) * tan(theta_h), (float)2.0);
            float bl = PI * pow(cos(theta_h), (float)4.0);
            float D = a_2 / (br * bl);

            //Shadow Masking term (Smith G Function)
            float G1_l = 0;
            if (dot(l, normal) > 0)
            {
                float theta_l = acos(dot(l, normal));
                G1_l = 2 / (1 + sqrt(1 + a_2 * pow(tan(theta_l), (float)2.0)));
            }
            float G1_v = 0;
            if (dot(-view, normal) > 0)
            {
                float theta_v = acos(dot(-view, normal));
                G1_v = 2 / (1 + sqrt(1 + a_2 * pow(tan(theta_v), (float)2.0)));
            }
            float G = G1_l * G1_v;

            //Fresnel term (Schlick's Approximation)
            float3 F = attrib.specular + (1 - attrib.specular) * pow(1 - clamp(dot(l, h), epsilon, (float)1), (float)5.0);
            //final output
            brdf_val = F * G * D/ (4 * clamp(dot(l, normal), epsilon, (float)1) * clamp(dot(-view, normal), epsilon, (float)1));
            brdf_val += attrib.diffuse / PI;
        }
    }
    float3 currWeight = pdf_inver * brdf_val * clamp(dot(l, normal), epsilon, (float)1);
    
    payload.radiance += 1.35 * result * payload.pathTracingWeight * boost;
    payload.pathTracingWeight *= currWeight;
    payload.origin = intersection;
    payload.dir = l;
}