#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_device.h>
#include <optixu/optixu_math_namespace.h>
#include "random.h"

#include "Payloads.h"
#include "Geometries.h"
#include "Light.h"

using namespace optix;

#define PI 3.1415926538

// Declare light buffers
rtBuffer<ParallelogramLight> qlights;
rtBuffer<PointLight> plights;
rtBuffer<DirectionalLight> dlights;

// Declare variables
rtDeclareVariable(Payload, payload, rtPayload, );
rtDeclareVariable(rtObject, root, , );
rtDeclareVariable(uint, lightsamples, , );
rtDeclareVariable(uint, lightstratify, , );

// Declare attibutes 
rtDeclareVariable(Attributes, attrib, attribute attrib, );
rtDeclareVariable(float3, intersection, attribute Intersection, );
rtDeclareVariable(float3, normal, attribute Normal, );
rtDeclareVariable(float3, view, attribute View, );
rtDeclareVariable(float, area, attribute Area, );

RT_PROGRAM void closestHit()
{
    float epsilon = 0.001f;
    float3 result = make_float3(0);
    //return the radiance if reach a light source
    if (abs(attrib.emission.x - 0.0) > epsilon || abs(attrib.emission.y - 0.0) > epsilon || abs(attrib.emission.z - 0.0) > epsilon)
    {
        if (payload.NEE == 0 || payload.first == 1)
        {
            payload.radiance += (attrib.emission / area) * payload.pathTracingWeight;
        }
        payload.done = true;
        return;
    }

    if (payload.NEE == 1)
    {
        for (int i = 0; i < qlights.size(); ++i)
        {
            ParallelogramLight qlight = qlights[i];
            float3 sum = make_float3(0, 0, 0);
            for (int j = 0; j < lightsamples; ++j)
            {
                float u1;
                float u2;
                if (lightstratify == 1)
                {
                    uint size = (uint)sqrt((float)lightsamples);
                    u1 = rnd(payload.seed) / size;
                    u2 = rnd(payload.seed) / size;
                    u1 += ((float)(j / size)) / size;
                    u2 += ((float)(j % size)) / size;
                }
                else
                {
                    u1 = rnd(payload.seed);
                    u2 = rnd(payload.seed);
                }
                //rtPrintf("%f %f \n", u1, u2);
                float3 xl = qlight.a + u1 * qlight.ab + u2 * qlight.ac;
                float3 l = normalize(xl - intersection);
                Ray shadowRay = make_Ray(intersection, l, 1, epsilon, length(xl - intersection) - epsilon);
                ShadowPayload shadowPayload;
                shadowPayload.isVisible = true;
                rtTrace(root, shadowRay, shadowPayload);
                if (!shadowPayload.isVisible)
                {
                    continue;
                }
                float3 r = 2 * normal * dot(normal, -view) + view;
                float3 brdf = attrib.diffuse / PI + attrib.specular * ((attrib.shininess + 2) / (2 * PI)) * pow(fmaxf(0.0, dot(r, l)), attrib.shininess);
                float costhetai = fmaxf(0.0, dot(normal, l));
                float3 nl = normalize(cross(qlight.ab, qlight.ac));
                float costhetao = fmaxf(0.0, dot(l, nl));
                float g = (costhetai * costhetao) / (length(xl - intersection) * length(xl - intersection));
                sum += brdf * g;
            }
            float A = length(cross(qlight.ab, qlight.ac));
            result += qlight.intensity * A * (sum / lightsamples);
        }
    }
    //indirect lighting
    float u1 = rnd(payload.seed);
    float u2 = rnd(payload.seed);
    float theta = acos(u1);
    float phi = 2 * PI * u2;
    float3 s = make_float3(cos(phi) * sin(theta), sin(theta) * sin(phi), cos(theta));
    float3 a;
    if (abs(normal.y - 1.0) < epsilon)
    {
        a = make_float3(1, 0, 0);
    }
    else
    {
        a = make_float3(0, 1, 0);
    }
    float3 u = normalize(cross(a, normal));
    float3 v = cross(normal, u);
    float3 l = s.x * u + s.y * v + s.z * normal;
    float3 r = 2 * normal * dot(normal, -view) + view;
    float3 brdf = attrib.diffuse / PI + attrib.specular * ((attrib.shininess + 2) / (2 * PI)) * pow(fmaxf(0.0, dot(r, l)), attrib.shininess);
    float3 currWeight = 2 * PI * brdf * dot(normal, l);
    
    payload.radiance += result * payload.pathTracingWeight;
    payload.pathTracingWeight *= currWeight;
    payload.origin = intersection;
    payload.dir = l;
    //payload.depth--;
}