#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_device.h>
#include <optixu/optixu_math_namespace.h>

#include "Payloads.h"

using namespace optix;

rtBuffer<float3, 2> resultBuffer; // used to store the render result

rtDeclareVariable(rtObject, root, , ); // Optix graph

rtDeclareVariable(uint2, launchIndex, rtLaunchIndex, ); // a 2d index (x, y)

rtDeclareVariable(int1, frameID, , );

// Camera info 

// TODO:: delcare camera varaibles here
rtDeclareVariable(uint, width, , );
rtDeclareVariable(uint, height, , );
rtDeclareVariable(float3, eye, , );
rtDeclareVariable(float3, center, , );
rtDeclareVariable(float3, up, , );
rtDeclareVariable(float, fov, , );

RT_PROGRAM void generateRays()
{
    float3 result = make_float3(0.f);

    // TODO: calculate the ray direction (change the following lines)
    float3 origin = eye; 
    float aspect_ratio = (float)width / (float)height;
    float3 camera_coord = make_float3(fov * aspect_ratio * (2.0 * ((launchIndex.x + 0.5) / width) - 1.0), fov * (2.0 * ((launchIndex.y + 0.5) / height) - 1.0), -1.0);
    //transform from screen space to world space
    Matrix<4, 4> camera_to_world;
    float3 z_axis = normalize(eye - center);
    float3 x_axis = normalize(cross(up, z_axis));
    camera_to_world.setCol(0, make_float4(x_axis.x, x_axis.y, x_axis.z, 0.0));
    camera_to_world.setCol(1, make_float4(up.x, up.y, up.z, 0.0));
    camera_to_world.setCol(2, make_float4(z_axis.x, z_axis.y, z_axis.z, 0.0));
    camera_to_world.setCol(3, make_float4(eye.x, eye.y, eye.z, 1.0));
    float4 p = camera_to_world * make_float4(camera_coord.x, camera_coord.y, camera_coord.z, 1.0);
    //get camera ray
    float3 dir = normalize(make_float3(p.x, p.y, p.z) - eye);
    float epsilon = 0.001f;

    // TODO: modify the following lines if you need
    // Shoot a ray to compute the color of the current pixel
    Ray ray = make_Ray(origin, dir, 0, epsilon, RT_DEFAULT_MAX);
    Payload payload;
    rtTrace(root, ray, payload);

    // Write the result
    resultBuffer[launchIndex] = payload.radiance;
}