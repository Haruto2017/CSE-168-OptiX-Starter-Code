#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_device.h>
#include <optixu/optixu_math_namespace.h>
#include "random.h"

#include "Payloads.h"

using namespace optix;

#define PI 3.1415926538

rtBuffer<float3, 2> resultBuffer; // used to store the render result

rtDeclareVariable(rtObject, root, , ); // Optix graph

rtDeclareVariable(uint2, launchIndex, rtLaunchIndex, ); // a 2d index (x, y)

rtDeclareVariable(int1, frameID, , );

// Camera info 

// TODO:: delcare camera varaibles here
rtDeclareVariable(uint, width, , );
rtDeclareVariable(uint, height, , );
rtDeclareVariable(float3, eye, , );
rtDeclareVariable(float3, center, , );
rtDeclareVariable(float3, up, , );
rtDeclareVariable(float, fov, , );
rtDeclareVariable(uint, spp, , );
rtDeclareVariable(uint, maxdepth, , );
rtDeclareVariable(uint, NEE, , );
rtDeclareVariable(uint, RR, , );
rtDeclareVariable(float, gamma, , );

RT_PROGRAM void generateRays()
{
    float3 result = make_float3(0.f);

    //rtPrintf("any\n");

    // TODO: calculate the ray direction (change the following lines)
    float aspect_ratio = (float)width / (float)height;

    float3 sum = make_float3(0);
    for (int i = 0; i < spp; ++i)
    {
        float3 origin = eye;
        size_t2 resultSize = resultBuffer.size();
        uint seed = tea<16>(launchIndex.x * resultSize.y + launchIndex.y, i);
        float3 camera_coord;
        if (i == 0)
        {
            camera_coord = make_float3(fov * aspect_ratio * (2.0 * ((launchIndex.x + 0.5) / width) - 1.0), fov * (2.0 * ((launchIndex.y + 0.5) / height) - 1.0), -1.0);
        }
        else
        {
            float u1 = rnd(seed);
            float u2 = rnd(seed);
            camera_coord = make_float3(fov * aspect_ratio * (2.0 * ((launchIndex.x + u1) / width) - 1.0), fov * (2.0 * ((launchIndex.y + u2) / height) - 1.0), -1.0);
        }
        //transform from screen space to world space
        Matrix<4, 4> camera_to_world;
        float3 z_axis = normalize(eye - center);
        float3 x_axis = normalize(cross(up, z_axis));
        camera_to_world.setCol(0, make_float4(x_axis.x, x_axis.y, x_axis.z, 0.0));
        camera_to_world.setCol(1, make_float4(up.x, up.y, up.z, 0.0));
        camera_to_world.setCol(2, make_float4(z_axis.x, z_axis.y, z_axis.z, 0.0));
        camera_to_world.setCol(3, make_float4(eye.x, eye.y, eye.z, 1.0));
        float4 p = camera_to_world * make_float4(camera_coord.x, camera_coord.y, camera_coord.z, 1.0);
        //get camera ray
        float3 dir = normalize(make_float3(p.x, p.y, p.z) - eye);
        float epsilon = 0.001f;
        // Shoot a ray to compute the color of the current pixel
        Payload payload;
        payload.done = false;
        payload.first = 1;
        if (NEE == 1)
        {
            payload.NEE = 1;
            payload.depth = maxdepth - 1;
        }
        else
        {
            payload.NEE = 0;
            payload.depth = maxdepth;
        }
        payload.RR = RR;
        payload.origin = origin;
        payload.dir = dir;
        payload.pathTracingWeight = make_float3(1.0);
        payload.radiance = make_float3(0);
        while (!payload.done && (payload.depth > 0 || RR))
        {
            payload.seed = tea<16>(i * resultSize.x * resultSize.y + launchIndex.x * resultSize.y + launchIndex.y, i * maxdepth + payload.depth);
            Ray ray = make_Ray(origin, dir, 0, epsilon, RT_DEFAULT_MAX);
            rtTrace(root, ray, payload);
            payload.first = 0;
            origin = payload.origin;
            dir = payload.dir;
            payload.depth--;
        }
        sum += payload.radiance;
        //rtPrintf("%d\n", payload.depth);
    }
    result = sum/spp;
    result = make_float3(pow(result.x, (float)(1 / gamma)), pow(result.y, (float)(1 / gamma)), pow(result.z, (float)(1 / gamma)));
    // Write the result
    resultBuffer[launchIndex] = result;
}