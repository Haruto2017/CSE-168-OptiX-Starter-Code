#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_device.h>
#include <optixu/optixu_math_namespace.h>
#include "random.h"

#include "Payloads.h"
#include "Geometries.h"
#include "Light.h"

using namespace optix;

// Declare light buffers
rtBuffer<PointLight> plights;
rtBuffer<DirectionalLight> dlights;

// Declare variables
rtDeclareVariable(Payload, payload, rtPayload, );
rtDeclareVariable(rtObject, root, , );

// Declare attibutes 
rtDeclareVariable(Attributes, attrib, attribute attrib, );
rtDeclareVariable(float3, intersection, attribute Intersection, );
rtDeclareVariable(float3, normal, attribute Normal, );
rtDeclareVariable(float3, view, attribute View, );

RT_PROGRAM void closestHit()
{
    // TDOO: calculate the color using the Blinn-Phong reflection model
    float3 result = attrib.emission + attrib.ambient;
    float epsilon = 0.001f;
    for (int i = 0; i < plights.size(); ++i)
    {
        PointLight point = plights[i];
        float3 l = normalize(point.position - intersection);
        float dis = length(point.position - intersection);
        Ray shadowRay = make_Ray(intersection, l, 1, epsilon, dis);
        ShadowPayload shadowPayload;
        shadowPayload.isVisible = true;
        rtTrace(root, shadowRay, shadowPayload);
        if (shadowPayload.isVisible)
        {
            float3 light = point.color / (point.attenuation.x + point.attenuation.y * dis + point.attenuation.z * dis * dis);
            float3 h = normalize(-view + l);
            result += attrib.diffuse * light * fmaxf(0.0, dot(normal, l)) + light * pow(fmaxf(0, dot(normal, h)), attrib.shininess);
        }
    }
    for (int i = 0; i < dlights.size(); ++i)
    {
        DirectionalLight dir = dlights[i];
        float3 l = dir.direction;
        Ray shadowRay = make_Ray(intersection, l, 1, epsilon, RT_DEFAULT_MAX);
        ShadowPayload shadowPayload;
        shadowPayload.isVisible = true;
        rtTrace(root, shadowRay, shadowPayload);
        if (shadowPayload.isVisible)
        {
            float3 h = normalize(-view + l);
            result += attrib.diffuse * dir.color * fmaxf(0.0, dot(normal, l)) + dir.color * pow(fmaxf(0, dot(normal, h)), attrib.shininess);
        }
    }

    payload.radiance = result;
}