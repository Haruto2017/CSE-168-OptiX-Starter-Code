#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_device.h>
#include "Geometries.h"

using namespace optix;

#define PI 3.1415926538

rtBuffer<Sphere> spheres; // a buffer of all spheres

rtDeclareVariable(Ray, ray, rtCurrentRay, );

// Attributes to be passed to material programs 
rtDeclareVariable(Attributes, attrib, attribute attrib, );
rtDeclareVariable(float3, intersection, attribute Intersection, );
rtDeclareVariable(float3, normal, attribute Normal, );
rtDeclareVariable(float3, view, attribute View, );
rtDeclareVariable(float, area, attribute Area, );

RT_PROGRAM void intersect(int primIndex)
{
    // Find the intersection of the current ray and sphere
    Sphere sphere = spheres[primIndex];
    float t;

    // TODO: implement sphere intersection test here
    float a = dot(ray.direction, ray.direction);
    float b = 2 * dot(ray.origin - sphere.center, ray.direction);
    float c = dot(ray.origin - sphere.center, ray.origin - sphere.center) - sphere.radius * sphere.radius;
    float delta = b * b - 4 * a * c;
    if (delta <= 0)
    {
        t = -1.0;
    }
    else
    {
        float t0 = (-b + sqrt(delta)) / (2 * a);
        float t1 = (-b - sqrt(delta)) / (2 * a);
        if (t0 > ray.tmin && t1 > ray.tmin)
        {
            t = (t0 < t1) ? t0 : t1;
        }
        else if (t0 > ray.tmin)
        {
            t = t0;
        }
        else
        {
            t = t1;
        }
    }

    // Report intersection (material programs will handle the rest)
    if (rtPotentialIntersection(t))
    {
        // Pass attributes

        // TODO: assign attribute variables here
        attrib.ambient = sphere.ambient;
        attrib.emission = sphere.emission;
        attrib.diffuse = sphere.diffuse;
        attrib.specular = sphere.specular;
        attrib.shininess = sphere.shininess;
        attrib.roughness = sphere.roughness;
        attrib.brdf = sphere.brdf;

        intersection = ray.origin + t * ray.direction;
        normal = normalize(intersection - sphere.center);
        view = ray.direction;
        area = 4 * PI * sphere.radius * sphere.radius;

        rtReportIntersection(0);
    }
}

RT_PROGRAM void bound(int primIndex, float result[6])
{
    Sphere sphere = spheres[primIndex];

    // TODO: implement sphere bouding box
    result[0] = -1000.f;
    result[1] = -1000.f;
    result[2] = -1000.f;
    result[3] = 1000.f;
    result[4] = 1000.f;
    result[5] = 1000.f;
}