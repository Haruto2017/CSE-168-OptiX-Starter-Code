#include "hip/hip_runtime.h"
#include "optix.h"
#include "optix_device.h"
#include "Geometries.h"

using namespace optix;

rtBuffer<Triangle> triangles; // a buffer of all spheres
rtBuffer<float3> vertices;

rtDeclareVariable(Ray, ray, rtCurrentRay, );

// Attributes to be passed to material programs 
rtDeclareVariable(Attributes, attrib, attribute attrib, );

RT_PROGRAM void intersect(int primIndex)
{
    // Find the intersection of the current ray and triangle
    Triangle tri = triangles[primIndex];
    float3 v0 = vertices[tri.vertice0];
    float3 v1 = vertices[tri.vertice1];
    float3 v2 = vertices[tri.vertice2];
    float t;
    //if(primIndex > 800)
        //rtPrintf("i: %d\n", primIndex);
    // TODO: implement triangle intersection test here
    float u;
    float v;
    float3 v0v1 = v1 - v0;
    float3 v0v2 = v2 - v0;
    float3 pvec = cross(ray.direction, v0v2);
    float det = dot(v0v1, pvec);
    // Backface Culling
    float epsilon = 0.0001f;
    if (det < epsilon)
    {
        t = -1;
    }
    else
    {
        float invDet = 1 / det;
        float3 tvec = ray.origin - v0;
        u = dot(tvec, pvec) * invDet;
        if (u < 0 || u > 1)
        {
            t = -1;
        }
        else
        {
            float3 qvec = cross(tvec, v0v1);
            v = dot(ray.direction, qvec) * invDet;
            if (v < 0 || u + v > 1)
            {
                t = -1;
            }
            else
            {
                t = dot(v0v2, qvec) * invDet;
            }
        }
    }

    // Report intersection (material programs will handle the rest)
    if (rtPotentialIntersection(t))
    {
        // Pass attributes

        // TODO: assign attribute variables here

        rtReportIntersection(0);
    }
}

RT_PROGRAM void bound(int primIndex, float result[6])
{
    Triangle tri = triangles[primIndex];

    // TODO: implement triangle bouding box
    result[0] = -1000.f;
    result[1] = -1000.f;
    result[2] = -1000.f;
    result[3] = 1000.f;
    result[4] = 1000.f;
    result[5] = 1000.f;
}