#include "hip/hip_runtime.h"
#include "optix.h"
#include "optix_device.h"
#include "Geometries.h"

using namespace optix;

rtBuffer<Triangle> triangles; // a buffer of all spheres
rtBuffer<float3> vertices;

rtDeclareVariable(Ray, ray, rtCurrentRay, );

// Attributes to be passed to material programs 
rtDeclareVariable(Attributes, attrib, attribute attrib, );
rtDeclareVariable(float3, intersection, attribute Intersection, );
rtDeclareVariable(float3, normal, attribute Normal, );
rtDeclareVariable(float3, view, attribute View, );
rtDeclareVariable(float, area, attribute Area, );

RT_PROGRAM void intersect(int primIndex)
{
    // Find the intersection of the current ray and triangle
    Triangle tri = triangles[primIndex];
    float t;
    //if(primIndex > 800)
        //rtPrintf("i: %d\n", primIndex);
    // TODO: implement triangle intersection test here
    float u;
    float v;
    float3 v0v1 = tri.vertice1 - tri.vertice0;
    float3 v0v2 = tri.vertice2 - tri.vertice0;
    float3 pvec = cross(ray.direction, v0v2);
    float det = dot(v0v1, pvec);
    // Backface Culling
    float epsilon = 0.001f;
    if (det < epsilon)
    {
        t = -1;
    }
    else
    {
        float invDet = 1 / det;
        float3 tvec = ray.origin - tri.vertice0;
        u = dot(tvec, pvec) * invDet;
        if (u < 0 || u > 1)
        {
            t = -1;
        }
        else
        {
            float3 qvec = cross(tvec, v0v1);
            v = dot(ray.direction, qvec) * invDet;
            if (v < 0 || u + v > 1)
            {
                t = -1;
            }
            else
            {
                t = dot(v0v2, qvec) * invDet;
            }
        }
    }

    // Report intersection (material programs will handle the rest)
    if (rtPotentialIntersection(t))
    {
        // Pass attributes

        // TODO: assign attribute variables here
        attrib.ambient = tri.ambient;
        attrib.emission = tri.emission;
        attrib.diffuse = tri.diffuse;
        attrib.specular = tri.specular;
        attrib.shininess = tri.shininess;

        intersection = ray.origin + t * ray.direction;
        normal = tri.normal;
        view = ray.direction;
        area = length(cross(v0v1, v0v2)) / 2;

        rtReportIntersection(0);
    }
}

RT_PROGRAM void bound(int primIndex, float result[6])
{
    Triangle tri = triangles[primIndex];

    // TODO: implement triangle bouding box
    result[0] = -1000.f;
    result[1] = -1000.f;
    result[2] = -1000.f;
    result[3] = 1000.f;
    result[4] = 1000.f;
    result[5] = 1000.f;
}